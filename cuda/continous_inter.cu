#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include "ATen/ATen.h"
typedef at::BFloat16 bf16;
typedef at::Half fp16;
typedef float fp32;

template <typename F>
__global__ void inter_fwd_kernel(const int B, const int T, const int C, const int H, const int CK, float *__restrict__ _state, int * __restrict__ _state_idx, int * __restrict__ _length, 
                               const F *__restrict__ const _r, const float *__restrict__ _w, F *__restrict__ const _y)
                               // CK : chunk nums
{
    const int b = blockIdx.x;
    const int ck= blockIdx.y+1;
    const int h = blockIdx.z;
    const int i = threadIdx.x;

    __shared__ float r[_N_], cw[_N_];
    float state[_N_];
    int t0 = _state_idx[ck*T-1]*H*_N_*_N_ + h*_N_*_N_ + i*_N_;
    #pragma unroll
    for (int j = 0; j < _N_; j++)
        state[j] = _state[t0 + j];
    cw[i]=1.0f;
    __syncthreads();

    // process y
    const int t_end = b*CK*T*C + ck*T*C + h*_N_ + i + _length[b*CK + ck]*C;
    for (int t = b*CK*T*C + ck*T*C + h*_N_ + i; t < t_end; t += C)
    {
        __syncthreads();
        r[i] = float(_r[t]);
        __syncthreads();
        float y = float(_y[t]);
        #pragma unroll
        for (int j = 0; j < _N_; j += 4)
        {
            const float4& r_ = (float4&)(r[j]);
            const float4& w_ = (float4&)(cw[j]);
            const float4& state_ = (float4&)(state[j]);
            y += r_.x*w_.x*state_.x;
            y += r_.y*w_.y*state_.y;
            y += r_.z*w_.z*state_.z;
            y += r_.w*w_.w*state_.w;
        }
        _y[t] = F(y);
        __syncthreads();
        cw[i] *= _w[t];
        __syncthreads();
    }
}
void inter_fwd_bf16(int B, int T, int C, int H, int CK, float *state, int *_state_idx, int *_length, bf16 *r, float *w, bf16 *y)
{
    assert(H*_N_ == C);
    inter_fwd_kernel<<<dim3(B, CK-1, H), dim3(_N_)>>>(B, T, C, H, CK, state, _state_idx, _length, r, w, y);
}
void inter_fwd_fp16(int B, int T, int C, int H, int CK, float *state, int *_state_idx, int *_length, fp16 *r, float *w, fp16 *y)
{
    assert(H*_N_ == C);
    inter_fwd_kernel<<<dim3(B, CK-1, H), dim3(_N_)>>>(B, T, C, H, CK, state, _state_idx, _length, r, w, y);
}
void inter_fwd_fp32(int B, int T, int C, int H, int CK, float *state, int *_state_idx, int *_length, fp32 *r, float *w, fp32 *y)
{
    assert(H*_N_ == C);
    inter_fwd_kernel<<<dim3(B, CK-1, H), dim3(_N_)>>>(B, T, C, H, CK, state, _state_idx, _length, r, w, y);
}